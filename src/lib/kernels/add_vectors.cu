#include "hip/hip_runtime.h"
#include "kernels/add_vectors.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

namespace learn_cuda
{

__global__
void VectorAddKernel(float const * fst, float const * snd, float * res, size_t size)
{
  size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < size)
    res[i] = fst[i] + snd[i];
}

void CudaVectorAdd(float const * fst, float const * snd, float * res, size_t size)
{
  size_t size_bytes = size * sizeof(float);
  float * d_fst, * d_snd, * d_res;
  hipMalloc(reinterpret_cast<void **>(&d_fst), size_bytes);
  hipMemcpy(d_fst, fst, size_bytes, hipMemcpyHostToDevice);
  hipMalloc(reinterpret_cast<void **>(&d_snd), size_bytes);
  hipMemcpy(d_snd, snd, size_bytes, hipMemcpyHostToDevice);
  hipMalloc(reinterpret_cast<void **>(&d_res), size_bytes);

  // run n/256 blocks of 256 threads each
  VectorAddKernel<<<ceil(size / 256.0), 256>>>(d_fst, d_snd, d_res, size);

  hipMemcpy(res, d_res, size_bytes, hipMemcpyDeviceToHost);
  hipFree(d_fst);
  hipFree(d_snd);
  hipFree(d_res);
}

}
